#include "hip/hip_runtime.h"
#include "grid.hpp"
#include <stdexcept>

using PlainCGrid = double*;

Grid::Grid(std::size_t grid_height, std::size_t grid_width): _grid_height(grid_height), _grid_width(grid_width) {
    auto success = hipMallocManaged(&_data, _grid_width * _grid_height * sizeof(double));
    if (success != hipSuccess) {
        throw std::runtime_error(hipGetErrorName(success));
    }
    for (int i = 0; i < _grid_width * _grid_height; i++) {
        _data[i] = 0;
    }
}

Grid::~Grid() {
    hipFree(_data);
}

#ifndef NO_CUDA
__global__
#endif
void cuda_step(const double* in, PlainCGrid out, std::size_t grid_width, std::size_t grid_height) {
    for(int i = 0; i < grid_height; i++) {
        out[i*GRID_WIDTH] = 1.0;
        for(int j = 1; j<grid_width; j++) {
            out[i*GRID_WIDTH + j] = in[i*GRID_WIDTH + j-1];
        }
    }
}

void World::step() {
#ifndef NO_CUDA
    cuda_step<<< 1, 1 >>>(current_grid->_data, other_grid->_data, other_grid->rows(), other_grid->cols());
#else
    cuda_step(current_grid->_data, other_grid->_data, other_grid->rows(), other_grid->cols());
#endif
    std::swap(other_grid, current_grid);
}
