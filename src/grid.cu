#include "hip/hip_runtime.h"
#include "grid.hpp"
#include <stdexcept>
#include <cassert>

template class Grid<double, 2>;

using PlainCGrid = double*;

template<class dtype, unsigned int dim>
Grid<dtype, dim>::Grid(std::array<const std::size_t, dim>&& dimensions):
    _size(std::move(dimensions)),
    GridView<dtype, dim>(nullptr, this->_size)
{
    auto success = hipMallocManaged(&this->_data, this->size() * sizeof(dtype));
    if (success != hipSuccess) {
        throw std::runtime_error(hipGetErrorName(success));
    }
    reset();
}

template<class dtype, unsigned int dim>
Grid<dtype, dim>::~Grid() {
    hipFree(this->_data);
}

void World::synchronize() {
#ifndef NO_CUDA
    hipDeviceSynchronize();
#endif
}

template<class dtype, unsigned int dim>
void Grid<dtype, dim>::reset() {
    memset(this->_data, 0, this->size() * sizeof(dtype));
}

constexpr double SINE_FREQ = 2.0; // in 1 / time unit
constexpr double SINE_FREQ_2PI = SINE_FREQ * 2 * M_PI;
constexpr double WAVE_SPEED = 0.5; // in space unit / time unit.

void nocuda_step(
    const double* in,
    PlainCGrid out,
    double t,
    double c, // C is the Courant number, c = v dx/dt. It must hold that 0 <= c < 1
    std::size_t grid_width, std::size_t grid_height
) {
    for(int i = 0; i < grid_height; i++) {
        out[i*grid_width] = sin(t * SINE_FREQ_2PI) + 1;
        for(int j = 1; j<grid_width; j++) {
            out[i*grid_width + j] =  in[i*grid_width + j] - c * (in[i*grid_width + j] - in[i*grid_width + j-1]);
        }
    }
}

__device__
inline void _cuda_step(
    const double* in,
    PlainCGrid out,
    double t,
    double c, // C is the Courant number, c = v dx/dt. It must hold that 0 <= c < 1
    std::size_t grid_width, std::size_t grid_height,
    std::size_t block_size
) {
    int start = threadIdx.x * block_size;
    int end = (threadIdx.x + 1) * block_size;
    for(int i = start; i < end; i++) {
        out[i*grid_width] = sin(t * SINE_FREQ_2PI) + 1;
        for(int j = 1; j<grid_width; j++) {
            out[i*grid_width + j] =  in[i*grid_width + j] - c * (in[i*grid_width + j] - in[i*grid_width + j-1]);
        }
    }
}

__global__ void cuda_step(PlainCGrid in, PlainCGrid out, double t, double c, std::size_t grid_width, std::size_t grid_height, std::size_t block_size) {
    _cuda_step(in, out, t, c, grid_width, grid_height, block_size);
}

__global__
void cuda_multistep(
    PlainCGrid in,
    PlainCGrid out,
    double t,
    double c,
    std::size_t grid_width, std::size_t grid_height,
    std::size_t block_size,
    unsigned N
) {
    for(unsigned i = 0; i < N; i++){
        _cuda_step(in, out, t, c, grid_width, grid_height, block_size);
	// can't use std::swap in device code, have to code our own
	auto tmp = in; in = out; out = tmp;
    }
}

void World::step() {
    const double c = WAVE_SPEED * grid1.shape()[0] * dt;
    assert(c <= 1.0);
#ifndef NO_CUDA
    cuda_step<<< 1, other_grid->shape()[1] >>>(current_grid->data(), other_grid->data(), t, c, other_grid->shape()[1], other_grid->shape()[0], 1);
#else
    nocuda_step(current_grid->_data, other_grid->_data, t, c, other_grid->rows(), other_grid->cols());
#endif
    std::swap(other_grid, current_grid);
    t += dt;
}

void World::multi_step(unsigned N) {
    const double c = WAVE_SPEED * grid1.shape()[0] * dt;
    assert(c <= 1.0);
#ifndef NO_CUDA
    cuda_multistep<<< 1, other_grid->shape()[1] >>>(current_grid->data(), other_grid->data(), t, c, other_grid->shape()[1], other_grid->shape()[0], 1, N);
#else
    for(unsigned i = 0; i < N; i++){
        nocuda_step(current_grid->_data, other_grid->_data, t, c, other_grid->rows(), other_grid->cols());
        std::swap(other_grid, current_grid);
    }
#endif
    t += N * dt;
}
