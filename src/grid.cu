#include "hip/hip_runtime.h"
#include "grid.hpp"
#include <stdexcept>
#include <cassert>

using PlainCGrid = double*;

Grid::Grid(std::size_t grid_height, std::size_t grid_width): _grid_height(grid_height), _grid_width(grid_width) {
    auto success = hipMallocManaged(&_data, _grid_width * _grid_height * sizeof(double));
    if (success != hipSuccess) {
        throw std::runtime_error(hipGetErrorName(success));
    }
    reset();
}

Grid::~Grid() {
    hipFree(_data);
}

void World::synchronize() {
#ifndef NO_CUDA
    hipDeviceSynchronize();
#endif
}

void Grid::reset() {
    memset(_data, 0, _grid_width * _grid_height * sizeof(*_data));
}

constexpr double SINE_FREQ = 2.0; // in 1 / time unit
constexpr double SINE_FREQ_2PI = SINE_FREQ * 2 * M_PI;
constexpr double WAVE_SPEED = 0.5; // in space unit / time unit.

void nocuda_step(
    const double* in,
    PlainCGrid out,
    double t,
    double c, // C is the Courant number, c = v dx/dt. It must hold that 0 <= c < 1
    std::size_t grid_width, std::size_t grid_height
) {
    for(int i = 0; i < grid_height; i++) {
        out[i*grid_width] = sin(t * SINE_FREQ_2PI) + 1;
        for(int j = 1; j<grid_width; j++) {
            out[i*grid_width + j] =  in[i*grid_width + j] - c * (in[i*grid_width + j] - in[i*grid_width + j-1]);
        }
    }
}

__device__
inline void _cuda_step(
    const double* in,
    PlainCGrid out,
    double t,
    double c, // C is the Courant number, c = v dx/dt. It must hold that 0 <= c < 1
    std::size_t grid_width, std::size_t grid_height,
    std::size_t block_size
) {
    int start = threadIdx.x * block_size;
    int end = (threadIdx.x + 1) * block_size;
    for(int i = start; i < end; i++) {
        out[i*grid_width] = sin(t * SINE_FREQ_2PI) + 1;
        for(int j = 1; j<grid_width; j++) {
            out[i*grid_width + j] =  in[i*grid_width + j] - c * (in[i*grid_width + j] - in[i*grid_width + j-1]);
        }
    }
}

__global__ void cuda_step(PlainCGrid in, PlainCGrid out, double t, double c, std::size_t grid_width, std::size_t grid_height, std::size_t block_size) {
    _cuda_step(in, out, t, c, grid_width, grid_height, block_size);
}

__global__
void cuda_multistep(
    PlainCGrid in,
    PlainCGrid out,
    double t,
    double c,
    std::size_t grid_width, std::size_t grid_height,
    std::size_t block_size,
    unsigned N
) {
    for(unsigned i = 0; i < N; i++){
        _cuda_step(in, out, t, c, grid_width, grid_height, block_size);
	// can't use std::swap in device code, have to code our own
	auto tmp = in; in = out; out = tmp;
    }
}

void World::step() {
    const double c = WAVE_SPEED * grid1.cols() * dt;
    assert(c <= 1.0);
#ifndef NO_CUDA
    cuda_step<<< 1, other_grid->cols() >>>(current_grid->_data, other_grid->_data, t, c, other_grid->rows(), other_grid->cols(), 1);
#else
    nocuda_step(current_grid->_data, other_grid->_data, t, c, other_grid->rows(), other_grid->cols());
#endif
    std::swap(other_grid, current_grid);
    t += dt;
}

void World::multi_step(unsigned N) {
    const double c = WAVE_SPEED * grid1.cols() * dt;
    assert(c <= 1.0);
#ifndef NO_CUDA
    cuda_multistep<<< 1, other_grid->cols() >>>(current_grid->_data, other_grid->_data, t, c, other_grid->rows(), other_grid->cols(), 1, N);
#else
    for(unsigned i = 0; i < N; i++){
        nocuda_step(current_grid->_data, other_grid->_data, t, c, other_grid->rows(), other_grid->cols());
        std::swap(other_grid, current_grid);
    }
#endif
    t += N * dt;
}
