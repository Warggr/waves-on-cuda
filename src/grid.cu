#include "hip/hip_runtime.h"
#include "grid.hpp"
#include <stdexcept>

using PlainCGrid = double*;

Grid::Grid() {
    auto success = hipMallocManaged(&_data, GRID_WIDTH * GRID_HEIGHT * sizeof(double));
    if (success != hipSuccess) {
        throw std::runtime_error(hipGetErrorName(success));
    }
    for (int i = 0; i < GRID_WIDTH * GRID_HEIGHT; i++) {
        _data[i] = 0;
    }
}

Grid::~Grid() {
    hipFree(_data);
}

__global__
void cuda_step(PlainCGrid in, PlainCGrid out) {
    for(int i = 0; i < GRID_HEIGHT; i++) {
        out[i*GRID_WIDTH] = 1.0;
        for(int j = 1; j<GRID_WIDTH; j++) {
            out[i*GRID_WIDTH + j] = in[i*GRID_WIDTH + j-1];
        }
    }
}

void World::step() {
    cuda_step<<< 1, 1 >>>(current_grid->_data, other_grid->_data );
    std::swap(other_grid, current_grid);
}
