#include "hip/hip_runtime.h"
#include "grid.hpp"
#include <stdexcept>
#include <cassert>

using PlainCGrid = double*;

Grid::Grid(std::size_t grid_height, std::size_t grid_width): _grid_height(grid_height), _grid_width(grid_width) {
    auto success = hipMallocManaged(&_data, _grid_width * _grid_height * sizeof(double));
    if (success != hipSuccess) {
        throw std::runtime_error(hipGetErrorName(success));
    }
    reset();
}

Grid::~Grid() {
    hipFree(_data);
}

void World::synchronize() {
#ifndef NO_CUDA
    hipDeviceSynchronize();
#endif
}

void Grid::reset() {
    memset(_data, 0, _grid_width * _grid_height * sizeof(*_data));
}

constexpr double SINE_FREQ = 2.0; // in 1 / time unit
constexpr double SINE_FREQ_2PI = SINE_FREQ * 2 * M_PI;
constexpr double WAVE_SPEED = 0.5; // in space unit / time unit.

void step(
    const double* in,
    PlainCGrid out,
    double t,
    double c, // C is the Courant number, c = v dx/dt. It must hold that 0 <= c < 1
    std::size_t grid_width, std::size_t grid_height
) {
    for(int i = 0; i < grid_height; i++) {
        out[i*grid_width] = sin(t * SINE_FREQ_2PI) + 1;
        for(int j = 1; j<grid_width; j++) {
            out[i*grid_width + j] =  in[i*grid_width + j] - c * (in[i*grid_width + j] - in[i*grid_width + j-1]);
        }
    }
}

__global__
void cuda_step(
    const double* in,
    PlainCGrid out,
    double t,
    double c, // C is the Courant number, c = v dx/dt. It must hold that 0 <= c < 1
    std::size_t grid_width, std::size_t grid_height,
    std::size_t block_size
) {
    int start = threadIdx.x * block_size;
    int end = (threadIdx.x + 1) * block_size;
    for(int i = start; i < end; i++) {
        out[i*grid_width] = sin(t * SINE_FREQ_2PI) + 1;
        for(int j = 1; j<grid_width; j++) {
            out[i*grid_width + j] =  in[i*grid_width + j] - c * (in[i*grid_width + j] - in[i*grid_width + j-1]);
        }
    }
}

void World::step(bool sync) {
    const double c = WAVE_SPEED * grid1.cols() * dt;
    assert(c <= 1.0);
#ifndef NO_CUDA
    cuda_step<<< 1, other_grid->cols() >>>(current_grid->_data, other_grid->_data, t, c, other_grid->rows(), other_grid->cols(), 1);
    if (sync) {
        hipDeviceSynchronize();
    }
#else
    step(current_grid->_data, other_grid->_data, t, c, other_grid->rows(), other_grid->cols());
#endif
    std::swap(other_grid, current_grid);
    t += dt;
}
